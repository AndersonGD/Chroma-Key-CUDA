#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "stb_image.h"
#include "stb_image_write.h"
using namespace std;

struct Image {
	unsigned char* data;
	int* dataCompressed;
	int width;
	int height;
	int nrChannels;
};

//Image host_Image;
//Image* dev_Image;

unsigned char* host_img;
int* host_imgCompressed;
int host_width;
int host_height;
int host_channels;

unsigned char* dev_img;
int* dev_imgCompressed;
int* dev_width;
int* dev_height;
int* dev_channels;

void LoadImage(const char* path, Image* img) {

	img->data = stbi_load(path, &img->width, &img->height, &img->nrChannels, 4);
}

void EmpacotarBits(Image* img) {
	int j = 0;

	//cout << "Empacotando bits de: " << img << endl;

	for (int i = 0; i < img->height * img->width * img->nrChannels; i += img->nrChannels)
	{
		int r = img->data[i];
		int g = img->data[i + 1];
		int b = img->data[i + 2];
		int a = img->data[i + 3];
		int rgba = (r << 24) | (g << 16) | (b << 8) | (a);
		img->dataCompressed[j] = rgba;
		//cout << rgba << endl;
		j++;
	}
}

void DesempacotarBits(Image* img) {

	int j = 0;

	for (int i = 0; i < img->height * img->width; i++)
	{
		int rgba = img->dataCompressed[i];
		int r1 = (rgba >> 24) & 0xff;
		int g1 = (rgba >> 16) & 0xff;
		int b1 = (rgba >> 8) & 0xff;
		int a1 = rgba & 0xff;

		img->data[j] = r1;
		img->data[j + 1] = g1;
		img->data[j + 2] = b1;
		img->data[j + 3] = a1;

		j += img->nrChannels;
	}
}

//#define cudaCheckErrors(msg) \
//    do { \
//        hipError_t __err = hipGetLastError(); \
//        if (__err != hipSuccess) { \
//            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
//                msg, hipGetErrorString(__err), \
//                __FILE__, __LINE__); \
//            fprintf(stderr, "*** FAILED - ABORTING\n"); \
//            exit(1); \
//        } \
//    } while (0)


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	//if (code != hipSuccess)
	//{
	fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	//if (abort) exit(code);
	//}
}

float DIST(unsigned char c0, unsigned char c1) {
	int r1 = (c0 >> 16) & 0xff;
	int g1 = (c0 >> 8) & 0xff;
	int b1 = c0 & 0xff;

	int r2 = (c1 >> 16) & 0xff;
	int g2 = (c1 >> 8) & 0xff;
	int b2 = c1 & 0xff;

	int r = r1 - r2;
	int g = g1 - g2;
	int b = b1 - b2;

	return sqrt(r * r + g * g + b * b);
}

unsigned char** ConvertFloatImageToUC(float* img, int width, int height) {
	int r, g, b, a, l;
	l = width * height;
	unsigned int c;
	unsigned char** char_buffer = new unsigned char*[width];
	for (int i = 0; i < width; i++)
	{
		char_buffer[i] = new unsigned char[height];
	}
	int index = 0;
	for (size_t i = 0; i < width; i++)
	{
		for (size_t j = 0; j < height; j++)
		{
			index = i + j * width;
			r = img[index] * 255;
			g = img[index + 1] * 255;
			b = img[index + 2] * 255;
			a = img[index + 3] * 255;
			c = a + (r << 16) + (g << 8) + b;

			char_buffer[i][j] = c;
		}
	}

	return char_buffer;
}

//__global__
//void ChromaKey(Image *img, int r, int g, int b)
//{
//	double red = 0.0, green = 0.0, blue = 0.0;
//	int index;
//	
//	index = threadIdx.x + blockIdx.x * img->width;
//	int pixel = img->dataCompressed[index];
//	int r1 = (pixel >> 24) & 0xff;
//	int g1 = (pixel >> 16) & 0xff;
//	int b1 = (pixel >> 8)  & 0xff;
//	int a1 = pixel & 0xff;
//	
//
//	red = r1 - r;
//	green = g1 - g;
//	blue = b1 - b;
//
//	float dist = sqrt(red * red + green * green + blue * blue);
//
//	if (dist <= 500) {
//		a1 = 0;
//		pixel = (r1 << 24 | g1 << 16 | b1 << 8 | a1);
//		img->dataCompressed[index] = pixel;
//	}
//}

__global__
void ChromaKey(int* img, int w, int h, int r, int g, int b)
{
	double red = 0.0, green = 0.0, blue = 0.0;
	int index;

	index = threadIdx.x + blockIdx.x * w;
	int pixel = img[index];
	int r1 = (pixel >> 24) & 0xff;
	int g1 = (pixel >> 16) & 0xff;
	int b1 = (pixel >> 8) & 0xff;
	int a1 = pixel & 0xff;


	red = r1 - r;
	green = g1 - g;
	blue = b1 - b;

	float dist = sqrt(red * red + green * green + blue * blue);

	if (dist <= 200) {
		a1 = 0;
		pixel = 0;// (r1 << 24 | g1 << 16 | b1 << 8 | a1);
		img[index] = pixel;
	}
}

void normalChroma(float* pixels, int i, int r, int g, int b) {
	double red = 0.0, green = 0.0, blue = 0.0;
	float r1 = pixels[i];
	float g1 = pixels[i + 1];
	float b1 = pixels[i + 2];

	red = r1 - (r / 255);
	green = g1 - (g / 255);
	blue = b1 - (b / 255);

	float dist = sqrt(red * red + green * green + blue * blue);

	if (dist <= 0.85f) {
		pixels[i + 3] = 0.0f;
	}

}

void init(void)
{
	//LoadImage("florest.jpg", &host_Image);
	host_img = stbi_load("colors.jpg", &host_width, &host_height, &host_channels, 4);
	host_channels = 4;

	int n = host_width * host_height * host_channels;

	host_imgCompressed = new int[host_width * host_height];

	int j = 0;
	//cout << "Empacotando bits de: " << img << endl;
	for (int i = 0; i < n; i += host_channels)
	{
		int r = host_img[i];
		int g = host_img[i + 1];
		int b = host_img[i + 2];
		int a = host_img[i + 3];
		int rgba = (r << 24) | (g << 16) | (b << 8) | (a);
		host_imgCompressed[j] = rgba;
		//cout << rgba << endl;
		j++;
	}
	int n2 = host_width * host_height;
	gpuErrchk(hipMallocManaged(&dev_img, (n * sizeof(unsigned char))));
	gpuErrchk(hipMallocManaged(&dev_imgCompressed, n2 * sizeof(int)));
	gpuErrchk(hipMallocManaged(&dev_channels, sizeof(int)));
	gpuErrchk(hipMallocManaged(&dev_height, sizeof(int)));
	gpuErrchk(hipMallocManaged(&dev_width, sizeof(int)));

	gpuErrchk(hipMemcpy(dev_img, host_img, n * sizeof(unsigned char), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_imgCompressed, host_imgCompressed, n2 * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_channels, &host_channels, sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_height, &host_height, sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dev_width, &host_width, sizeof(int), hipMemcpyHostToDevice));

	ChromaKey << <*dev_width, *dev_height >> > (dev_imgCompressed, *dev_width, *dev_height, 0, 255, 0);

	gpuErrchk(hipMemcpy(host_imgCompressed, dev_imgCompressed, n2 * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	j = 0;

	for (int i = 0; i < host_height * host_width; i++)
	{
		int rgba = host_imgCompressed[i];
		int r1 = (rgba >> 24) & 0xff;
		int g1 = (rgba >> 16) & 0xff;
		int b1 = (rgba >> 8) & 0xff;
		int a1 = rgba & 0xff;

		host_img[j] = r1;
		host_img[j + 1] = g1;
		host_img[j + 2] = b1;
		host_img[j + 3] = a1;

		j += host_channels;
	}
	stbi_write_png("test.png", host_width, host_height, 4, host_img, host_width * 4);


	//EmpacotarBits(&host_Image);

	/*int dSize = host_Image.width * host_Image.height * host_Image.nrChannels;
	int dcSize = host_Image.width * host_Image.height;*/

	//gpuErrchk(hipMallocManaged(&dev_Image, (dSize + dcSize) * sizeof(Image)));
	//cudaCheckErrors("cudaMalloc1 fail");
	////dev_Image->dataCompressed = host_Image.dataCompressed;

	////gpuErrchk(hipMemcpy(dev_Image, &host_Image, sizeof(Image), hipMemcpyHostToDevice));
	//cudaCheckErrors("cudaMalloc1 fail");
	//for (int i = 0; i < dSize; i++) {
	//	hipMalloc((void**) & (host_Image.data[i]), 300 * sizeof(unsigned char));
	//	//cudaCheckErrors("cudaMalloc2 fail");
	//	hipMemcpy(&dev_Image->data[i], &host_Image.data[i], sizeof(unsigned char*), hipMemcpyHostToDevice);
	//	//cudaCheckErrors("cudaMemcpy1 fail");
	//}
	//gpuErrchk(hipMemcpy(dev_width, &host_width, 512 * sizeof(int), hipMemcpyHostToDevice));
	//gpuErrchk(hipMemcpy(dev_height, &host_height, 512 * sizeof(int), hipMemcpyHostToDevice));

	//ChromaKey << <dev_Image->width, dev_Image->height >> > (dev_Image, 255, 0, 0);


	/*gpuErrchk(hipMemcpy(&host_Image, dev_Image, sizeof(Image), hipMemcpyDeviceToHost));
	DesempacotarBits(&host_Image);
	stbi_write_png("test.png", host_Image.width, host_Image.height, 4, host_Image.data, 0);*/

	/*int l = host_height * host_width *STBI_rgb_alpha;
	for (int i = 0; i < l; i+=4)
	{
	normalChroma(host_ImageData, i, 0, 255, 0);
	}
	*/
	/*hipMemcpy(host_ImageData, dev_pixels,512 * sizeof(unsigned char), hipMemcpyDeviceToHost);*/


	//save
	//unsigned char** buffer = ConvertFloatImageToUC(host_ImageData, host_width, host_height);

	//l = host_width * host_height * 4;
	//unsigned char* char_buffer = new unsigned char[l];

	//for (int i = 0; i < host_width * host_height * 4; i += 4) {
	//	char_buffer[i] = host_ImageData[i] * 255;
	//	char_buffer[i+1] = host_ImageData[i + 1] * 255;
	//	char_buffer[i+2] = host_ImageData[i + 2] * 255;
	//	char_buffer[i+3] = host_ImageData[i + 3] * 255;
	//}

	//stbi_write_png("test.png", host_width, host_height, 4, char_buffer, host_width*4);
}



int main(int argc, char** argv)
{

	/*gpuErrchk(hipMallocManaged(&dev_pixels, 512 * sizeof(unsigned char)));
	gpuErrchk(hipMallocManaged(&dev_width, 512 * sizeof(int)));
	gpuErrchk(hipMallocManaged(&dev_height, 512 * sizeof(int)));*/

	init();

	//hipFree(reg)
	system("PAUSE");

	return 0;
}
